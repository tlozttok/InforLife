#include "hip/hip_runtime.h"
#include "Background.cuh"
#include "Function.cuh"
#include <cstdio>
typedef float (*f2f)(float);

__device__ int pos2offset(int x, int y, int c, int size)
{
	int s = 0;
	int c_size = size * size;
	s += c_size * c;
	s += size * y;
	s += x;
	return s;
}

__device__ int3 offset2pos(int offset, int size)
{
	int c_size = size * size;
	int3 p = int3();
	p.x = offset % c_size % size;
	p.y = int((offset % c_size) / size);
	p.z = int(offset / c_size);
	return p;
}

__device__ void conv(float* data, float* kernel, float* sum, int k_size, int channel, int x, int y, int d_size, float* result)//������
{
	int d_offset;
	int k_offset = 0;
	int r;
	for (int c = 0; c < channel; c++) {
		r = 0;
		for (int iy = y - k_size; iy < y + k_size; iy++) {
			d_offset = pos2offset(x - k_size, iy, c, d_size);
			for (int ix = x - k_size; ix < x + k_size; ix++) {
				if (ix >= 0 && ix < d_size && iy >= 0 && iy < d_size) {
					r += data[d_offset] * kernel[k_offset];
				}
				d_offset++;
				k_offset++;
			}
		}
		result[c] = r / sum[c];
	}
}

__device__ void activate(float* data, int channel, f2f af) {
	for (int i = 0; i < channel; i++) {
		data[i] = af(data[i]);
	}
}

__device__ void matmul(float* data, float* mat, int width, int height, float* result) {
	int count = 0;
	int r;
	for (int iy = 0; iy < height; iy++) {
		r = 0;
		for (int ix = 0; ix < width; ix++) {
			r += mat[count] * data[ix];
			count++;
		}
		result[iy] = r;
	}
}

__device__ void respond(float* data, int channel, ActionPair* RF) {
	for (int i = 0; i < channel; i++) {
		data[i] = mix_gaussian_gpu(data[i], RF);
	}
}


__global__ void step_compute(
	float* data,
	gene** gene_mask, 
	float* dynamic, 
	float delta_t,
	bool* action_mask, 
	float* data_b, 
	float* data_d, 
	float* n_data,
	int size, 
	int channel
) {
	int id = threadIdx.x;
	int num = size * size;
	int amount = int(num / blockDim.x + 1);
	for (int i = id * amount; i < (id+1) * amount; i++) {
		if (i < num) {
			int3 pos = offset2pos(i, size);
			gene* g = gene_mask[i];
			float* conv_r = 0;
			hipMalloc((void**)&conv_r, sizeof(float) * channel);//���������ڴ�
			printf("i: %d \n", i);
			conv(data, g->conv_kernel, g->kernel_sum, g->k_length, channel, pos.x, pos.y, size, conv_r);//���
			printf("i: %d \n", i);
			activate(conv_r, channel, sin_af);//����
			printf("i: %d \n", i);
			float* mat_r = 0;
			hipMalloc((void**)&mat_r, sizeof(float) * channel);//����任������ڴ�
			matmul(conv_r, g->FCL_matrix, size, size, mat_r);//����˷�
			respond(mat_r, channel, &(g->step));//�õ����
			printf("i: %d \n", i);
			for (int c = 0; c < channel; c++) {
				n_data[i + c * num] = mat_r[c] * delta_t + dynamic[i + c * num];
			}
			if (action_mask[i]) {
				float born = 0;
				float death = 0;
				matmul(conv_r, g->weight, channel, 1, &born);//ϸ����������
				death = born;
				respond(&born, 1, &(g->born));
				respond(&death, 1, &(g->death));
				death -= g->limit;
				born -= g->limit;
				data_b[i] = born;
				data_d[i] = death;
			}
		}
	}
}

Env::Env(int size, int channel, Cells* cells):size(size),channel(channel),cells(cells)
{
	cudaStatus = hipMalloc((void**)&data, sizeof(float) * size * size * channel);
	cudaStatus = hipMalloc((void**)&data_b, sizeof(float) * size * size);
	cudaStatus = hipMalloc((void**)&data_d, sizeof(float) * size * size);
	cudaStatus = hipMalloc((void**)&gene_mask, sizeof(gene*) * size * size);
	cudaStatus = hipMalloc((void**)&dynamic, sizeof(float) * size * size);
	cudaStatus = hipMalloc((void**)&action_mask, sizeof(bool) * size * size);
	hipMemset(data, 0, sizeof(float) * size * size * channel);
	hipMemset(data_b, 0, sizeof(float) * size * size);
	hipMemset(data_d, 0, sizeof(float) * size * size);
	hipMemset(gene_mask, 0, sizeof(gene*) * size * size);
	hipMemset(dynamic, 0, sizeof(float) * size * size);
	hipMemset(action_mask, 0, sizeof(bool) * size * size);
}

Env::~Env()
{
	cudaStatus = hipFree(data);
	cudaStatus = hipFree(data_b);
	cudaStatus = hipFree(data_d);
	cudaStatus = hipFree(gene_mask);
	cudaStatus = hipFree(dynamic);
	cudaStatus = hipFree(action_mask);
}

void Env::step()
{
	if (cell_territory_lock.try_lock())//���Զ�ȡ����
	{
		gene** genemask = cells->get_gene_mask();
		hipMemcpy(gene_mask, genemask, sizeof(gene*) * size * size,hipMemcpyHostToDevice);
		hipMemcpy(action_mask, cells->get_action_mask(), sizeof(bool) * size * size, hipMemcpyHostToDevice);
		cell_territory_lock.unlock();
	}
	if (dynamic_lock.try_lock())//���Զ�ȡ����
	{
		hipMemcpy(dynamic, cells->get_dynamic(), sizeof(float) * size * size, hipMemcpyHostToDevice);
		dynamic_lock.unlock();
	}
	float* ndata = 0;//�����ݱ����Թ������̶߳�ȡ
	float* n_data_b = 0;
	float* n_data_d = 0;
	cudaStatus = hipMallocManaged((void**)&ndata, sizeof(float) * size * size * channel);
	cudaStatus = hipMallocManaged((void**)&n_data_b, sizeof(float) * size * size);
	cudaStatus = hipMallocManaged((void**)&n_data_d, sizeof(float) * size * size);
	step_compute<<<1,2>>>(data, gene_mask, dynamic, delta_t, action_mask, n_data_b, n_data_d, ndata, size, channel);
	hipDeviceSynchronize();
	gpu_data_lock.lock();//��ָ�д����
	hipFree(data);
	hipFree(data_b);
	hipFree(data_d);
	data = ndata;
	data_b = n_data_b;
	data_d = n_data_d;
	gpu_data_lock.unlock();
}

int* Env::get_data_img()
{
	float* fdata = new float[sizeof(float) * size * size * channel];
	int* img_data = new int[sizeof(float) * size * size * channel];
	hipMemcpy(fdata, data, sizeof(float) * size * size * channel, hipMemcpyDeviceToHost);
	for (int i = 0; i < size * size * channel; i++) {
		img_data[i] = int(256 * cut(fdata[i]));
	}
	return img_data;
}